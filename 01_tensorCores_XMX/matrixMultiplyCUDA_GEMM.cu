//==============================================================
// Copyright © 2023 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================
// haowei.zhang@intel.com

// HPCSDK: nvc++ -acc -mp=gpu -gpu=cc70 -Minfo=all -I/usr/local/cuda/targets/x86_64-linux/include -L/usr/local/cuda/lib64 -lcublas matrixMultiplyCUDA_GEMM.cu
// CUDA Toolkit: nvcc -arch=sm_70 -lcublas matrixMultiplyCUDA_GEMM.cu


#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cmath>
#include <cstring>
#include <hipblas.h>

#define __TIME_BEGIN hipEventRecord(start);
#define __TIME_END              \
    hipEventRecord(stop);      \
    hipEventSynchronize(stop); \
    hipEventElapsedTime(&elapsedTime, start, stop);

// #define cudaErrCheck(stat)                         \
//     {                                              \
//         cudaErrCheck_((stat), __FILE__, __LINE__); \
//     }
// void cudaErrCheck_(cudaError_t stat, const char *file, int line)
// {
//     if (stat != cudaSuccess)
//     {
//         fprintf(stderr, "CUDA Error: %s %s %d\n", cudaGetErrorString(stat), file, line);
//     }
// }

// #define cublasErrCheck(stat)                         \
//     {                                                \
//         cublasErrCheck_((stat), __FILE__, __LINE__); \
//     }
// void cublasErrCheck_(cublasStatus_t stat, const char *file, int line)
// {
//     if (stat != CUBLAS_STATUS_SUCCESS)
//     {
//         fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
//     }
// }

// #define curandErrCheck(stat)                         \
//     {                                                \
//         curandErrCheck_((stat), __FILE__, __LINE__); \
//     }
// void curandErrCheck_(curandStatus_t stat, const char *file, int line)
// {
//     if (stat != CURAND_STATUS_SUCCESS)
//     {
//         fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
//     }
// }

#define TILE_WIDTH 16
#define N_REPEAT 100

typedef float fp;

static int M = 2048;
static int K = 1024;
static int N = 512;

fp *arrayA_h, *arrayB_h, *arrayC_h, *arrayC_href;
fp *arrayA_d, *arrayB_d, *arrayC_d;
hipEvent_t start, stop;

float elapsedTime;

void resources_init();
void result_reset();
void resources_free();
void print_matrix(const fp *arr, int M, int N);
bool compare_matrix(const fp *arr1, const fp *arr2, int M, int N, bool transpose = false);
void multiplyCpu(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpu(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuSh(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuShBc(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuShBcPd(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuAcc(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuOmp(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);
void multiplyGpuGemm(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N);

int main()
{
    resources_init();

    multiplyCpu(arrayA_h, arrayB_h, arrayC_href, M, K, N);

    multiplyGpu(arrayA_d, arrayB_d, arrayC_d, M, K, N);

    multiplyGpuSh(arrayA_d, arrayB_d, arrayC_d, M, K, N);

    multiplyGpuShBc(arrayA_d, arrayB_d, arrayC_d, M, K, N);

    multiplyGpuShBcPd(arrayA_d, arrayB_d, arrayC_d, M, K, N);

    multiplyGpuAcc(arrayA_h, arrayB_h, arrayC_h, M, K, N);

    multiplyGpuOmp(arrayA_h, arrayB_h, arrayC_h, M, K, N);

    multiplyGpuGemm(arrayA_d, arrayB_d, arrayC_d, M, K, N);

    resources_free();
    return 0;
}

void resources_init()
{
    arrayA_h = new fp[M * K]();
    arrayB_h = new fp[K * N]();
    arrayC_h = new fp[M * N]();
    arrayC_href = new fp[M * N]();

    hipMalloc((void **)&arrayA_d, M * K * sizeof(fp));
    hipMalloc((void **)&arrayB_d, K * N * sizeof(fp));
    hipMalloc((void **)&arrayC_d, M * N * sizeof(fp));

    for (int i = 0; i < M * K; i++)
        arrayA_h[i] = rand() / (fp)RAND_MAX * 1.0;

    for (int i = 0; i < K * N; i++)
        arrayB_h[i] = rand() / (fp)RAND_MAX * 1.0;

    memset(arrayC_h, 0, M * N * sizeof(fp));
    memset(arrayC_href, 0, M * N * sizeof(fp));

    hipMemcpy(arrayA_d, arrayA_h, M * K * sizeof(fp), hipMemcpyHostToDevice);
    hipMemcpy(arrayB_d, arrayB_h, K * N * sizeof(fp), hipMemcpyHostToDevice);
    hipMemset(arrayC_d, 0, M * N * sizeof(fp));
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

void result_reset()
{
    memset(arrayC_h, 0, M * N * sizeof(fp));
    hipMemset(arrayC_d, 0, M * N * sizeof(fp));
}

void resources_free()
{
    delete arrayA_h;
    delete arrayB_h;
    delete arrayC_h;
    delete arrayC_href;
    hipFree(arrayA_d);
    hipFree(arrayB_d);
    hipFree(arrayC_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void print_matrix(const fp *arr, int M, int N)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << arr[i * N + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

bool compare_matrix(const fp *arr1, const fp *arr2, int M, int N, bool transpose)
{
    fp err = 0;
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (!transpose)
                err = fabs(arr1[i * N + j] - arr2[i * N + j]);
            else
                err = fabs(arr1[j * M + i] - arr2[i * N + j]);
            if (err > fabs(arr2[i * N + j] * 1.E-4))
            {
                printf("   Mismatch at i = %d, j = %d, err = %f, arr2 = %f\n", i, j, err, arr2[i * N + j]);
                // if (!transpose)
                //     print_matrix(arr1, M, N);
                // else
                //     print_matrix(arr1, N, M);
                // print_matrix(arr2, M, N);
                return false;
            }
        }
    }
    return true;
}

void multiplyCpu(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    __TIME_BEGIN
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < K; k++)
            {
                arrC[i * N + j] += arrA[i * K + k] * arrB[k * N + j];
            }
        }
    }
    __TIME_END
    printf("0. CPU calculation time = %f ms\n", elapsedTime);
    // print_matrix(arrA, M, K);
    // print_matrix(arrB, K, N);
    // print_matrix(arrC, M, N);
}

__global__ void _matrixMul(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    // absolute row and col
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;
    for (int k = 0; k < K; k++)
    {
        arrC[row * N + col] += arrA[row * K + k] * arrB[k * N + col];
    }
}

void multiplyGpu(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    __TIME_BEGIN
    _matrixMul<<<dimGrid, dimBlock>>>(arrA, arrB, arrC, M, K, N);
    __TIME_END

    hipMemcpy(arrayC_h, arrC, M * N * sizeof(fp), hipMemcpyDeviceToHost);
    if (!compare_matrix(arrayC_h, arrayC_href, M, N))
    {
        std::cout << "1. Error at multiplyGpu" << std::endl;
    }
    else
    {
        printf("1. Pass, GPU calculation time (without shared memory) = %f ms\n", elapsedTime);
    }
}

__global__ void _matrixMulSh(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    // absolute row and col
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ fp arrAs[TILE_WIDTH * TILE_WIDTH];
    __shared__ fp arrBs[TILE_WIDTH * TILE_WIDTH];
    fp elementC = 0;

    for (int i = 0; i < K / TILE_WIDTH; i++)
    {
        arrAs[threadIdx.y * TILE_WIDTH + threadIdx.x] = arrA[row * K + i * TILE_WIDTH + threadIdx.y];
        arrBs[threadIdx.y * TILE_WIDTH + threadIdx.x] = arrB[(i * TILE_WIDTH + threadIdx.x) * N + col];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
            elementC += arrAs[k * TILE_WIDTH + threadIdx.x] * arrBs[threadIdx.y * TILE_WIDTH + k];
        __syncthreads();
    }
    arrC[row * N + col] = elementC;
}

void multiplyGpuSh(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    __TIME_BEGIN
    _matrixMulSh<<<dimGrid, dimBlock>>>(arrA, arrB, arrC, M, K, N);
    __TIME_END

    hipMemcpy(arrayC_h, arrC, M * N * sizeof(fp), hipMemcpyDeviceToHost);
    if (!compare_matrix(arrayC_h, arrayC_href, M, N))
    {
        std::cout << "2. Error at multiplyGpuSh" << std::endl;
    }
    else
    {
        printf("2. Pass, GPU calculation time (with shared memory) = %f ms\n", elapsedTime);
    }
}

__global__ void _matrixMulShBc(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    // absolute row and col
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ fp arrAs[TILE_WIDTH * TILE_WIDTH];
    __shared__ fp arrBs[TILE_WIDTH * TILE_WIDTH];
    fp elementC = 0;

    for (int i = 0; i < K / TILE_WIDTH; i++)
    {
        arrAs[threadIdx.x * TILE_WIDTH + threadIdx.y] = arrA[row * K + i * TILE_WIDTH + threadIdx.y];
        arrBs[threadIdx.x * TILE_WIDTH + threadIdx.y] = arrB[(i * TILE_WIDTH + threadIdx.x) * N + col];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
            elementC += arrAs[threadIdx.x * TILE_WIDTH + k] * arrBs[k * TILE_WIDTH + threadIdx.y];
        __syncthreads();
    }
    arrC[row * N + col] = elementC;
}

void multiplyGpuShBc(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    __TIME_BEGIN
    _matrixMulShBc<<<dimGrid, dimBlock>>>(arrA, arrB, arrC, M, K, N);
    __TIME_END

    hipMemcpy(arrayC_h, arrC, M * N * sizeof(fp), hipMemcpyDeviceToHost);
    if (!compare_matrix(arrayC_h, arrayC_href, M, N))
    {
        std::cout << "3. Error at multiplyGpuShBc" << std::endl;
    }
    else
    {
        printf("3. Pass, GPU calculation time (with shared memory, bank conflict) = %f ms\n", elapsedTime);
    }
}

__global__ void _matrixMulShBcPd(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    // absolute row and col
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ fp arrAs[TILE_WIDTH * (TILE_WIDTH + 1)];
    __shared__ fp arrBs[TILE_WIDTH * (TILE_WIDTH + 1)];
    fp elementC = 0;

    for (int i = 0; i < K / TILE_WIDTH; i++)
    {
        arrAs[threadIdx.x * (TILE_WIDTH + 1) + threadIdx.y] = arrA[row * K + i * TILE_WIDTH + threadIdx.y];
        arrBs[threadIdx.x * (TILE_WIDTH + 1) + threadIdx.y] = arrB[(i * TILE_WIDTH + threadIdx.x) * N + col];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
            elementC += arrAs[threadIdx.x * (TILE_WIDTH + 1) + k] * arrBs[k * (TILE_WIDTH + 1) + threadIdx.y];
        __syncthreads();
    }
    arrC[row * N + col] = elementC;
}

void multiplyGpuShBcPd(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((M + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    __TIME_BEGIN
    _matrixMulShBcPd<<<dimGrid, dimBlock>>>(arrA, arrB, arrC, M, K, N);
    __TIME_END

    hipMemcpy(arrayC_h, arrC, M * N * sizeof(fp), hipMemcpyDeviceToHost);
    if (!compare_matrix(arrayC_h, arrayC_href, M, N))
    {
        std::cout << "4. Error at multiplyGpuShBcPd" << std::endl;
    }
    else
    {
        printf("4. Pass, GPU calculation time (with shared memory, bank conflict fixed with padding) = %f ms\n", elapsedTime);
    }
}

void multiplyGpuAcc(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
#pragma acc kernels
    {
        ;
    }
#pragma acc enter data copyin(arrA [0:M * K], arrB [0:K * N], arrC [0:M * N])
    __TIME_BEGIN
#pragma acc kernels default(present)
#pragma acc loop independent collapse(2)
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < K; k++)
            {
                arrC[i * N + j] += arrA[i * K + k] * arrB[k * N + j];
            }
        }
    }
    __TIME_END

#pragma acc update host(arrC [0:M * N])
#pragma acc exit data delete (arrA [0:M * K], arrB [0:K * N], arrC [0:M * N])
    if (!compare_matrix(arrC, arrayC_href, M, N))
    {
        std::cout << "5. Error at multiplyCpuAcc" << std::endl;
    }
    else
    {
        printf("5. Pass, GPU calculation time (OpenACC) = %f ms\n", elapsedTime);
    }
}

void multiplyGpuOmp(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
#pragma omp target
    {
        ;
    }
#pragma omp target enter data map(to \
                                  : arrA [0:M * K], arrB [0:K * N], arrC [0:M * N])
    __TIME_BEGIN
#pragma omp target teams distribute parallel for collapse(2)
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < K; k++)
            {
                arrC[i * N + j] += arrA[i * K + k] * arrB[k * N + j];
            }
        }
    }
    __TIME_END

#pragma omp target update from(arrC [0:M * N])
#pragma omp target exit data map(delete \
                                 : arrA [0:M * K], arrB [0:K * N], arrC [0:M * N])
    if (!compare_matrix(arrC, arrayC_href, M, N))
    {
        std::cout << "6. Error at multiplyCpuOmp" << std::endl;
    }
    else
    {
        printf("6. Pass, GPU calculation time (OpenMP) = %f ms\n", elapsedTime);
    }
}

// ref: https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/
//      https://blog.csdn.net/u011197534/article/details/78378536
void multiplyGpuGemm(const fp *arrA, const fp *arrB, fp *arrC, int M, int K, int N)
{
    result_reset();
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    // Use tensor cores
    hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
    bool transpose = false;
    __TIME_BEGIN
    // Now using cuBLAS
    // transpose = true;
    // cublasGemmEx(cublasHandle, CUBLAS_OP_T, CUBLAS_OP_T,
    //                             M, N, K,
    //                             &alpha,
    //                             arrA, CUDA_R_32F, K,
    //                             arrB, CUDA_R_32F, N,
    //                             &beta,
    //                             arrC, CUDA_R_32F, M,
    //                             CUDA_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
    transpose = false;
    for (int i = 0; i < N_REPEAT; i++)
    {
        hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     N, M, K,
                     &alpha,
                     arrB, HIP_R_32F, N,
                     arrA, HIP_R_32F, K,
                     &beta,
                     arrC, HIP_R_32F, N,
                     HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
    }
    __TIME_END
    hipMemcpy(arrayC_h, arrC, M * N * sizeof(fp), hipMemcpyDeviceToHost);
    if (!compare_matrix(arrayC_h, arrayC_href, M, N, transpose))
    {
        std::cout << "7. Error at multiplyGpuGemm" << std::endl;
    }
    else
    {
        printf("7. Pass, GPU calculation time (Gemm Tensor Cores) = %f ms\n", elapsedTime/N_REPEAT);
    }
    hipblasDestroy(cublasHandle);
}